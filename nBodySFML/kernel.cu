#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "thrust\device_vector.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>

#include "SFML\Graphics.hpp"
#include "Node.h"
#include <ctime>
#include <math.h>
#include <cmath>
#include <vector>
#include <omp.h>
#include <thread>
#include <random>
#include <chrono>


#define _PI 3.14159265      //Pi, used for calculations and rounded to 8 decimal places. 
#define _GRAV_CONST 0.1     //the gravitational constant. This is the timestep between each frame. Lower for slower but more accurate simulations

void BodyAttraction(std::vector<Body*> &pBodies, float pSoftener);                                                                       //Attracts each body to each other body in the given vector of pointers to body objects
void CalculateForceNode(Body* bi, Node* bj, float pSoftener);                                                                           //Calculate force exerted on body from node
void CalculateForce(Body* bi, Body* bj, float pSoftener);                                                                               //Calculate force exerted on eachother between two bodies
//Body* CreateBody(float px, float py, float pmass, float pvx = 0, float pvy = 0);                                                        //return a pointer to new body object defined on the heap with given paramiters
void DeleteBodies(std::vector<Body*> &pBodies);                                                                                          //Deletes objects pointed to by given vector
void PollEvent(sf::RenderWindow* pTarget, bool* pIsPaused, sf::View* pSimView);                                                         //Call all polled events for the sf::window
void PopulateBodyVectorDisk(std::vector<Body*> *pBodies, float pWidth, float pHeight, float pMaxDist, float pMinDist, float pMinMass, float pMaxMass, float pGalaticCenterMass = 0);  //populate given vector with bodies with given paramiters in a disk formation
void Render(sf::RenderWindow* pTarget, std::vector<Body*> pBodies, sf::Color pObjColor);                                                //Render given body objects to given screen
void SetView(sf::View* pView, sf::RenderWindow* pTarget, float pViewWidth, float pViewHeight);                                          //set the window to the simulation view
void UpdateBodies(std::vector<Body*> &pBodies);                                                                                          //Calculate velocity chance from the bodies force exerted since last update, update position based on velocity, reset force to 0
void DrawNode(Node* pNode, sf::RenderWindow* pTarget);                                                                                  //Draw a node to the screen, and all of its children (recursive)
void CheckNode(Node* pNode, Body* pBody);                                                                                               //Checks if a node is sufficently far away for force calculation, if not recureses on nodes children
void OctreeBodyAttraction();                                                                                                            //Using a calculated oct-tree, calculate the force exerted on each object
void AttractToCenter(std::vector<Body*> &pBodies, float width, float height, float centerMass);                                                                                                                  //Attract each particle to the center of the simulation
void ResetForces(std::vector<Body*> &pBodies);
void RepellFromCenter(std::vector<Body*> &pBodies, float width, float height, float centerMass);

float const DiskRadiusMax = 20000;              //Max and min distances objects will be from the galatic center
float const DiskRadiusMin = 50;
float const GalaticCenterMass = 1000000;        //The mass of the very large object simulating a black hole at the center of a galixy;
float const ObjectMassMax = 10;                  //The max and min mass of the objects in the galixy
float const ObjectMassMin = 1;
float const SimWidth = 327680;                  //Width and height of simulation, needs to be large, particles outside of this range will not be included in the octree
float const SimHeight = 327680;
float const ViewWidth = 1920;                   //Width and height of view of the simulation for the screen. 
float const ViewHeight = 1080;
float const Softener = 10;                      //A softener used for the force calculations, 10 is a good amount
#define pParticlesCount 10000       //Number of particles in simtulation, currently 2^15                                
double const _NODE_THRESHOLD = 0.5;             //Threshold for node calculations   

float zoom = 1;                                 //The current amount of zoom in or out the user has inputed in total
float MouseX = 0;
float MouseY = 0;

std::vector<Body*> Bodies;                      //Container of all Bodies in simulation
Node GlobalNode;
bool IsPaused = false;                          //Contains the state of weather the simulation is paused or not
sf::Color ObjColor(255, 255, 255, 128);         //the defult colour of the objects
sf::View SimulationView;
sf::RenderWindow window(sf::VideoMode(1920, 1080), "N-Body simulation");

int main()
{
	// N Body Code
	try
	{
		PopulateBodyVectorDisk(&Bodies, SimWidth, SimHeight, DiskRadiusMax, DiskRadiusMin, ObjectMassMin, ObjectMassMax, GalaticCenterMass);
	}
	catch (const std::exception &exc)
	{
		std::cerr << exc.what();
	}
	SetView(&SimulationView, &window, ViewWidth, ViewHeight);

	sf::Clock clock;
	float lastTime = 0;
	int frames = 0;
	float avgFps = 0;

	while (window.isOpen())
	{
		PollEvent(&window, &IsPaused, &SimulationView); //These will always be done

		if (!IsPaused)  //These will not if the simulation is paused
		{
			AttractToCenter(Bodies, SimWidth, SimHeight, GalaticCenterMass);
			UpdateBodies(Bodies);
			ResetForces(Bodies);
			GlobalNode.Reset();
			GlobalNode.SetParam(Bodies, SimWidth, SimHeight);
			OctreeBodyAttraction();
		}

		Render(&window, Bodies, ObjColor);
		
		float currentTime = clock.restart().asSeconds();
		float fps = 1.f / currentTime;
		avgFps += fps;
		++frames;
		
		lastTime = currentTime;

	}

	std::cout << "avg: " << avgFps / frames << std::endl;


	DeleteBodies(Bodies);
	// --------------------------------

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus;

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

    return 0;
}

void AttractToCenter(std::vector<Body *> &pBodies, float width, float height, float centerMass)
{
	Body* Temp = new Body(width / 2, height / 2, centerMass); //Create a body at the center of the simulation

	for (unsigned int i = 0; i < pBodies.size(); i++)
	{
		CalculateForce(pBodies[i], Temp, Softener);
	}

	delete Temp;
}

void RepellFromCenter(std::vector<Body*> &pBodies, float width, float height, float centerMass)
{
	Body* Temp = new Body(width / 2, height / 2, centerMass); //Create a body at the center of the simulation

	for (unsigned int i = 0; i < pBodies.size(); i++)
	{
		float vectorx = Temp->posX - pBodies[i]->posX;
		float vectory = Temp->posY - pBodies[i]->posY;

		float distSqr = vectorx * vectorx + vectory * vectory;

		double Dist = (sqrt(distSqr));

		double force = (pBodies[i]->mass * Dist * _GRAV_CONST * 0.0001);

		pBodies[i]->forceX -= vectorx * force;
		pBodies[i]->forceY -= vectory * force;
	}

	delete Temp;
}

void ResetForces(std::vector<Body*> &pBodies)
{
	for (unsigned int i = 0; i < pBodies.size(); i++)
	{
		pBodies[i]->forceX = 0;
		pBodies[i]->forceY = 0;
	}
}

void BodyAttraction(std::vector<Body*> &pBodies, float pSoftener)
{
	for (unsigned int i = 0; i < pBodies.size(); i++)
	{
		for (unsigned int j = 0; j < pBodies.size(); j++)
		{
			CalculateForce(pBodies.at(i), pBodies.at(j), pSoftener); //for each body in pBodies: each other body in pBodies: Calculate attractive force exerted on the first body from the second one
		}
	}
}

void OctreeBodyAttraction()
{

	for (unsigned int i = 0; i < Bodies.size(); i++)
	{
		CheckNode(&GlobalNode, Bodies[i]);
	}
}

inline void CheckNode(Node* pNode, Body* pBody)
{
	if (pNode->Bodies.size() != 0)
	{
		float diffX = (pNode->CenterOfMassx - pBody->posX);
		float diffY = (pNode->CenterOfMassy - pBody->posY);

		float distance = sqrt((diffX) * (diffX)+(diffY) * (diffY));   //Distance from the node to the object                          

		if ((pNode->width / distance) < (_NODE_THRESHOLD) || (pNode->HasChildren == false))     //if sufficently far away or has no children (external node) group node and calculate force
		{
			CalculateForceNode(pBody, pNode, Softener);
			pNode->IsUsed = true;
		}
		else                                                                                    //if not, repeat function with children
		{
			CheckNode(pNode->Child[0], pBody);
			CheckNode(pNode->Child[1], pBody);
			CheckNode(pNode->Child[2], pBody);
			CheckNode(pNode->Child[3], pBody);
		}
	}
}

inline void CalculateForceNode(Body* bi, Node* bj, float pSoftener)  //bi is being attracted to bj. 15 flops of calculation
{
	//vector from the body to the center of mass
	float vectorx = bj->CenterOfMassx - bi->posX;
	float vectory = bj->CenterOfMassy - bi->posY;

	//c^2 = a^2 + b^2 + softener^2
	float distSqr = vectorx * vectorx + vectory * vectory + pSoftener * pSoftener;

	// ivnDistCube = 1/distSqr^(3/2)
	float distSixth = distSqr * distSqr * distSqr;
	double invDistCube = 1.0f / (sqrt(distSixth));


	double force = (bj->TotalMass * bi->mass * invDistCube * _GRAV_CONST);

	bi->forceX += vectorx * force;
	bi->forceY += vectory * force;
}

//__global__ void calculateForceKernel(Body )

inline void CalculateForce(Body* bi, Body* bj, float pSoftener)  //bi is being attracted to bj. 15 flops of calculation
{
	//std::vector from i to j
	float vectorx = bj->posX - bi->posX;
	float vectory = bj->posY - bi->posY;

	//c^2 = a^2 + b^2 + softener^2
	float distSqr = vectorx * vectorx + vectory * vectory + pSoftener * pSoftener;

	// ivnDistCube = 1/distSqr^(3/2)
	float distSixth = distSqr * distSqr * distSqr;
	double invDistCube = 1.0f / (sqrt(distSixth));


	double force = (bj->mass * bi->mass * invDistCube * _GRAV_CONST);

	bi->forceX += vectorx * force;
	bi->forceY += vectory * force;
}

void DeleteBodies(std::vector<Body*> &pBodies)
{
	for (unsigned int i = 0; i < pBodies.size(); i++)
	{
		delete pBodies[i];
	}

	pBodies.clear();
}

void PollEvent(sf::RenderWindow* pTarget, bool* pIsPaused, sf::View* pSimView)
{
	sf::Event event;

	while (pTarget->pollEvent(event))
	{
		if (event.type == sf::Event::Closed)
			pTarget->close();
		if (event.type == sf::Event::KeyPressed)
		{
			if (event.key.code == sf::Keyboard::Space)
				*pIsPaused = !*pIsPaused;                   //toggle what is pointed to by IsPaused
		}
		if (event.type == sf::Event::MouseWheelScrolled)
		{
			zoom *= 1 + (static_cast <float> (-event.mouseWheelScroll.delta) / 10); //for each notch down, -10%, for each notch up, +10%
			pSimView->zoom(1 + (static_cast <float> (-event.mouseWheelScroll.delta) / 10));
		}
	}

	if (sf::Mouse::getPosition().x > (1920 - 20))
		SimulationView.move(2 * zoom, 0);
	if (sf::Mouse::getPosition().x < (0 + 20))
		SimulationView.move(-2 * zoom, 0);
	if (sf::Mouse::getPosition().y >(1080 - 20))
		SimulationView.move(0, 2 * zoom);
	if (sf::Mouse::getPosition().y < (0 + 20))
		SimulationView.move(0, -2 * zoom);

	pTarget->setView(*pSimView);
}

__global__ void setup_curand(hiprandState_t *states)
{
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	hiprand_init(0, idx, 0, &states[idx]);
}

__global__ void calculateMass(hiprandState_t *states, float *minMass, float *maxMass, float *varsArray)
{
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	varsArray[threadIdx.x] = *minMass + (hiprand_uniform(&states[idx]) * (*maxMass));
}

__global__ void calculatePosX(hiprandState_t *states, float *minDist, float *maxDist, float *width, float *varsArray)
{
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	float theta = hiprand_uniform(&states[idx]) * 2 * 3.14159265;
	float distanceCoefficient = hiprand_uniform(&states[idx]);

	float distance = *minDist + ((*maxDist - *minDist) * (distanceCoefficient * distanceCoefficient));

	varsArray[idx] = cos(theta) * distance + (*width / 2);
}

__global__ void calculatePosY(hiprandState_t *states, float *minDist, float *maxDist, float *height, float *varsArray)
{
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	float theta = hiprand_uniform(&states[idx]) * 2 * 3.14159265;
	float distanceCoefficient = hiprand_uniform(&states[idx]);

	float distance = *minDist + ((*maxDist - *minDist) * (distanceCoefficient * distanceCoefficient));

	varsArray[idx] = sin(theta) * distance + (*height / 2);
}

__global__ void calculateVelocityX(hiprandState_t *states, float *minDist, float *maxDist, float *galacticCenterMass, float *varsArray)
{
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	float theta = hiprand_uniform(&states[idx]) * 2 * 3.14159265;
	float distanceCoefficient = hiprand_uniform(&states[idx]);

	float distance = *minDist + ((*maxDist - *minDist) * (distanceCoefficient * distanceCoefficient));
	float orbitalVelocity = sqrt((*galacticCenterMass * static_cast <float> (_GRAV_CONST)) / distance);                  //Calculate the orbital velocity required to orbit the galatic centre   

	varsArray[idx] = (sin(theta) * orbitalVelocity);
}

__global__ void calculateVelocityY(hiprandState_t *states, float *minDist, float *maxDist, float *galacticCenterMass, float *varsArray)
{
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	float theta = hiprand_uniform(&states[idx]) * 2 * 3.14159265;
	float distanceCoefficient = hiprand_uniform(&states[idx]);

	float distance = *minDist + ((*maxDist - *minDist) * (distanceCoefficient * distanceCoefficient));
	float orbitalVelocity = sqrt((*galacticCenterMass * static_cast <float> (_GRAV_CONST)) / distance);                  //Calculate the orbital velocity required to orbit the galatic centre   

	varsArray[idx] = (-cos(theta) * orbitalVelocity);
}

void PopulateBodyVectorDisk(std::vector<Body*> *pBodies, float pWidth, float pHeight, float pMaxDist, float pMinDist, float pMinMass, float pMaxMass, float pGalaticCenterMass)
{
	auto start = std::chrono::system_clock::now();

	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<float> dist(0, 1); //pMinDist, pMaxDist);
	std::uniform_real_distribution<float> angle(0, 2*_PI);
	std::uniform_real_distribution<float> mass(pMinMass, pMaxMass);
	

	// cuda stuff
	float *massArray;
	float *posXArray;
	float *posYArray;
	float *velocityXArray;
	float *velocityYArray;
	float *width;
	float *height;
	float *minMass;
	float *maxMass;
	float *minDist;
	float *maxDist;
	float *galacticCenterMass;

	hiprandState_t *states;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	cudaStatus = hipMalloc((void**)&states, sizeof(hiprandState_t) * pParticlesCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&width, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&height, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&minMass, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&maxMass, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&minDist, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&maxDist, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&galacticCenterMass, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&massArray, sizeof(float) * pParticlesCount);	// this array will contain all the values needed to create all the particles
																// it is multiplied by 5 because there will be 5 variables for each Body
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&posXArray, sizeof(float) * pParticlesCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&posYArray, sizeof(float) * pParticlesCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&velocityXArray, sizeof(float) * pParticlesCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&velocityYArray, sizeof(float) * pParticlesCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(width, &pWidth, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy width failed!");
	}
	cudaStatus = hipMemcpy(height, &pHeight, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy height failed!");
	}
	cudaStatus = hipMemcpy(minMass, &pMinMass, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy minMass failed!");
	}
	cudaStatus = hipMemcpy(maxMass, &pMaxMass, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy maxMass failed!");
	}
	cudaStatus = hipMemcpy(minDist, &pMinDist, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy minDist failed!");
	}
	cudaStatus = hipMemcpy(maxDist, &pMaxDist, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy maxDist failed!");
	}
	cudaStatus = hipMemcpy(galacticCenterMass, &pGalaticCenterMass, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy galactic failed!");
	}

	setup_curand <<<pParticlesCount/1024, 1024 >>> (states);

	calculateMass <<<pParticlesCount/1024,1024>>> (states, minMass, maxMass, massArray);
	calculatePosX <<<pParticlesCount / 1024, 1024 >>> (states, minDist, maxDist, width, posXArray);
	calculatePosY <<<pParticlesCount / 1024, 1024 >>> (states, minDist, maxDist, height, posYArray);
	calculateVelocityX <<<pParticlesCount / 1024, 1024 >>> (states, minDist, maxDist, galacticCenterMass, velocityXArray);
	calculateVelocityY <<<pParticlesCount / 1024, 1024 >>> (states, minDist, maxDist, galacticCenterMass, velocityYArray);

	std::vector<float> massResults(pParticlesCount);
	std::vector<float> posXResults(pParticlesCount);
	std::vector<float> posYResults(pParticlesCount);
	std::vector<float> velocityXResults(pParticlesCount);
	std::vector<float> velocityYResults(pParticlesCount);

	hipMemcpy(&massResults[0], massArray, sizeof(float) * pParticlesCount, hipMemcpyDeviceToHost);
	hipMemcpy(&posXResults[0], posXArray, sizeof(float) * pParticlesCount, hipMemcpyDeviceToHost);
	hipMemcpy(&posYResults[0], posYArray, sizeof(float) * pParticlesCount, hipMemcpyDeviceToHost);
	hipMemcpy(&velocityXResults[0], velocityXArray, sizeof(float) * pParticlesCount, hipMemcpyDeviceToHost);
	hipMemcpy(&velocityYResults[0], velocityYArray, sizeof(float) * pParticlesCount, hipMemcpyDeviceToHost);

	// create the bodies
	for (int i = 0; i < pParticlesCount; i++) {
		//std::cout << posXResults[i] << " " << posYResults[i] << " " << massResults[i] << " " << velocityXResults[i] << " " << velocityYResults[i] << std::endl;
		float theta = angle(gen);
		float distanceCoefficient = dist(gen);

		float distance = pMinDist + ((pMaxDist - pMinDist) * (distanceCoefficient * distanceCoefficient));                    //Distance point will be from the galatic center, between MinDiskRadius and MaxDiskRadius

		float positionx = cos(theta) * distance + (pWidth / 2);                                                             //set positionx and positiony to be the point you get when you go in the direction of 'angle' till you have traveled 'distance' 
		float positiony = sin(theta) * distance + (pHeight / 2);

		float orbitalVelocity = sqrt((pGalaticCenterMass * static_cast <float> (_GRAV_CONST)) / distance);                  //Calculate the orbital velocity required to orbit the galatic centre   

		float velocityx = (sin(theta) * orbitalVelocity);
		float velocityy = (-cos(theta) * orbitalVelocity);

		//std::cout << positionx << " " << positiony << " " << mass(gen) << " " << velocityx << " " << velocityy << std::endl;

		pBodies->push_back(new Body(positionx, positiony, mass(gen), velocityx, velocityy));
		//pBodies->push_back(new Body(posXResults[i], posYResults[i], mass(gen)));
	}

	hipFree(massArray);
	hipFree(posXArray);
	hipFree(posYArray);
	hipFree(velocityXArray);
	hipFree(velocityYArray);
	hipFree(width);
	hipFree(height);
	hipFree(minMass);
	hipFree(maxMass);
	hipFree(minDist);
	hipFree(maxDist);
	hipFree(galacticCenterMass);
	hipFree(states);

	auto end = std::chrono::system_clock::now();
	auto total = end - start;
	std::cout << "Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(total).count() << std::endl;
}


void Render(sf::RenderWindow* pTarget, std::vector<Body*> pBodies, sf::Color pObjColor)
{
	pTarget->clear();

	sf::RectangleShape Temp;
	//Temp.setFillColor(pObjColor);

	for (unsigned int i = 0; i < pBodies.size(); i++)
	{
		if (zoom > 1)
			Temp.setSize(sf::Vector2f(pBodies.at(i)->mass * zoom, pBodies.at(i)->mass * zoom));
		else
			Temp.setSize(sf::Vector2f(pBodies.at(i)->mass, pBodies.at(i)->mass));

		float ForceCoiffecent = sqrt(pBodies.at(i)->forceX * pBodies.at(i)->forceX + pBodies.at(i)->forceY * pBodies.at(i)->forceY) * (40000 * _GRAV_CONST);

		if (ForceCoiffecent > 1)
			ForceCoiffecent = 1;

		float Red, Green, Blue;

		Blue = 1 - (ForceCoiffecent);

		if (ForceCoiffecent < 0.2)
			Red = (ForceCoiffecent) * 5;
		else
			Red = 1;

		if (ForceCoiffecent < 0.5)
			Green = (ForceCoiffecent) * 2;
		else
			Green = 1;

		Temp.setFillColor(sf::Color(Red * 255, Green * 255, Blue * 255, 128));
		Temp.setPosition(pBodies.at(i)->posX, pBodies.at(i)->posY);
		pTarget->draw(Temp);
	}

	//DrawNode(&GlobalNode, pTarget);

	pTarget->display();
}

void DrawNode(Node* pNode, sf::RenderWindow* pTarget)
{
	sf::RectangleShape Temp;
	Temp.setFillColor(sf::Color(0, 0, 0, 0));
	Temp.setOutlineThickness(zoom);
	Temp.setOutlineColor(sf::Color(0, 255, 0, 16));
	Temp.setPosition(pNode->posX, pNode->posY);
	Temp.setSize(sf::Vector2f(pNode->width, pNode->height));

	pTarget->draw(Temp);
	if (pNode->HasChildren) //recercivly draw all children
	{
		DrawNode(pNode->Child[0], pTarget);
		DrawNode(pNode->Child[1], pTarget);
		DrawNode(pNode->Child[2], pTarget);
		DrawNode(pNode->Child[3], pTarget);
	}
}

void SetView(sf::View* pView, sf::RenderWindow* pTarget, float pViewWidth, float pViewHeight)
{
	pView->reset(sf::FloatRect(SimWidth / 2 - pViewWidth / 2, SimHeight / 2 - pViewHeight / 2, pViewWidth, pViewHeight));
	pView->setViewport(sf::FloatRect(0.f, 0.f, 1.f, 1.f));
	pTarget->setView(*pView);
}


void UpdateBodies(std::vector<Body*> &pBodies)
{

	for (unsigned int i = 0; i < pBodies.size(); i++)
	{
		if ((pBodies[i]->posX > SimWidth && pBodies[i]->velX > 0) || (pBodies[i]->posX < 0 && pBodies[i]->velX < 0))
		{
			pBodies[i]->velX = -pBodies[i]->velX;
		}

		if ((pBodies[i]->posY > SimHeight && pBodies[i]->velY > 0) || (pBodies[i]->posY < 0 && pBodies[i]->velY < 0))
		{
			pBodies[i]->velY = -pBodies[i]->velY;
		}

		pBodies.at(i)->velX += pBodies.at(i)->forceX / pBodies.at(i)->mass;     //f = ma => force = mass * accelleration. Therefor
		pBodies.at(i)->velY += pBodies.at(i)->forceY / pBodies.at(i)->mass;     //a = f/m => accelleration = force / mass

		pBodies.at(i)->posX += pBodies.at(i)->velX;
		pBodies.at(i)->posY += pBodies.at(i)->velY;
	}
}